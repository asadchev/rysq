#include "hip/hip_runtime.h"
#include "rysq/rysq.h"
#include "rysq/roots/roots.h"
#include "rysq/roots/generated.h"

namespace rysq {
namespace cuda {

  struct Shell {
    int L;
    int K;
    RYSQ_GPU_ENABLED
    const auto& operator[](int idx) const {
      return orbitals_[idx];
    }
    struct Primitive {
      double a,C;
    };
    Primitive prims[20];
  private:
    rysq::Shell::Orbital orbitals_[RYSQ_MAX_CART];
  };


  RYSQ_GPU_ENABLED
  inline int nbf(const Shell &s) {
    return shell::nbf(s.L);
  }

  RYSQ_GPU_ENABLED
  inline int nprims(const Shell &s) {
    return s.K;
  }

  template<int N, int K, int P, class Orbitals>
  __device__
  void inner_kernel(const Orbitals &p, const double *Ix, const double *Iy, const double *Iz, double (&G)[P]) {

#define Ix(k,a,p) Ix[a + k*N + K*N*(p.x)]
#define Iy(k,a,p) Iy[a + k*N + K*N*(p.y)]
#define Iz(k,a,p) Iz[a + k*N + K*N*(p.z)]

    int k = threadIdx.x%K;

#pragma unroll 1
    for (int i = 0; i < P; ++i) {
      double gi = 0;
#pragma unroll 1
      for (int a = 0; a < N; ++a) {
        gi += Ix(a,k,p[i])*Iy(a,k,p[i])*Iz(a,k,p[i]);
      }
      G[i] += gi;
      //g[i] += Ix(1,k,p)*Iy(1,k,p)*Iz(1,k,p);
    }

  }

  template<int N, int PL>
  __global__
  void kernel(const Shell &P, const Shell &Q, double *G) {

    constexpr int NP = ((PL+1)*(PL+2))/2;
    constexpr int K = (32/NP);

    __shared__ double I[3][N*K*(PL+1)];
    __shared__ shell::Primitives primitives[32/NP];

    __shared__ double gp[NP];

    if (threadIdx.x < 30) {
      for (int q = threadIdx.y; q < nbf(Q); q += blockDim.y) {
        inner_kernel<N,K,NP>(P, I[0], I[1], I[2], gp);
      }
    }

    __syncthreads();

    for (int i = threadIdx.x; i < NP; i += blockDim.x) {
      G[i] = gp[i];
    }

  }

  // template
  // __global__
  // void kernel<2,4>(const Shell &P, const Shell &Q, double *G);

  template<int N>
  RYSQ_GPU_ENABLED
  double contract(const double *Ix, const double *Iy, const double *Iz) {
    double g = 0;
    #pragma unroll 1
    for (int i = 0; i < N; ++i) {
      g += Ix[i*3]*Iy[i*3]*Iz[i*3];
    }
    return g;
  }

  template<int N>
  RYSQ_GPU_ENABLED
  inline void recurrence(
    int m, int n,
    const double &A, const double &B,
    double rAB, double rAi, double rBk,
    double t2,
    double *G)
  {
    // bra recurrence G(a,m,0)
#define G(i) G[(i)*N]
    double C = (rAi - rAB*B*t2);
    G(0) = 1.0;
    G(1) = C;
    double A2 = 1.0/(2*A);
    double B1 = (1.0 - B*t2)*A2;
    #pragma unroll 1
    for (int i = 1; i < m; ++i) {
      G(i+1) = C*G(i) + double(i)*B1*G(i-1);
    }
#undef G
  }


  template<int N, int B>
  __global__
  void kernel_2c(const Shell &P, const Shell &R, double *G) {

    __shared__ double ri[3], rk[3];
    __shared__ double X[N], W[N];
    extern __shared__ double I[];

    const int thread = threadIdx.x+threadIdx.y*blockDim.x;
    double g[B] = { 0 };

#define I(i,j,x) (I + x + 3*N*(i + j*(P.L+1)))

    for (int k = 0; k < nprims(R); ++k) {
      for (int i = 0; i < nprims(P); ++i) {
        // compute I
        {
          if (thread < N) {
            double x = P.prims[i].a;
            roots3(x, X, W, thread);
          }
          __syncthreads();
          if (thread < 3*N) {
            int x = thread%3;
            int a = thread/3;
            double ai = P.prims[i].a;
            double ak = R.prims[k].a;
            double rA = ri[x];
            double rB = rk[x];
            double rAB = rA - rB;
            double rAi = 0;//rA - ri[x]; // 0?
            double rBk = 0;//rB - rk[x]; // 0?
            recurrence<N*3>(P.L, R.L, ai, ak, rAB, rAi, rBk, X[a], &I[x+a*3]);
          }
        }
        __syncthreads();
        double C = (P.prims[i].C*R.prims[k].C);
        auto p = P[threadIdx.x];
        //#pragma unroll
        for (int b = 0; b < B; ++b) {
          auto r = R[threadIdx.y+b*blockDim.y];
          double gb = C*contract<N>(I(p.x,r.x,0), I(p.y,r.y,1), I(p.z,r.z,2));
          //int idx = threadIdx.x + threadIdx.y*blockDim.x + b*(blockDim.x*blockDim.y);
          g[b] += gb;
        }
      }
    }

    __syncthreads();

    //#pragma unroll 1
    for (int b = 0; b < B; ++b) {
      int idx = threadIdx.x + threadIdx.y*blockDim.x + b*(blockDim.x*blockDim.y);
      G[idx] = g[b];
    }

  }

  template
  __global__
  void kernel_2c<3,3>(const Shell &P, const Shell &Q, double *G);


}
}
